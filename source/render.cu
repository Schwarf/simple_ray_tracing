#include "hip/hip_runtime.h"
//
// Created by andreas on 01.11.21.
//

#include "cuda_implementation/miscellaneous/templates/c_vector.h"
#include <fstream>

#define checkCudaErrors(value) check_cuda( (value), #value, __FILE__, __LINE__)

__global__ void render_it(c_vector3 *buffer, size_t max_width, size_t max_height)
{
	//size_t width = threadIdx.x + blockIdx.x * blockDim.x;
	//size_t height = threadIdx.y + blockIdx.y * blockDim.y;
	size_t width = threadIdx.x;
	size_t height = blockIdx.x;
	if ((width >= max_width) || (height >= max_height)) {
		return;
	}
	size_t pixel_index = height * max_width + width;
	buffer[pixel_index] = c_vector3{0.2, 0.7, 0.8};
}

int main()
{
	size_t width = 1024;
	size_t height = 768;
	// Why is 32 the maximum number of threads per block
	constexpr size_t threads_per_block = 32;
	//dim3 number_of_threads(threads_per_block, threads_per_block);

	//dim3 number_of_blocks(width / threads_per_block, height / threads_per_block);
	int number_of_blocks = 768;
	int number_of_threads{1024};
	size_t buffer_size = width * height * sizeof(float3);
	std::cout << buffer_size << std::endl;
	c_vector3 *buffer;
	hipMallocManaged((void **)&buffer, buffer_size);

	render_it<<<number_of_blocks, number_of_threads>>>(buffer, width, height);
	hipGetLastError();
	hipDeviceSynchronize();
	std::ofstream ofs;
	ofs.open("./cuda_image.ppm");
	ofs << "P6\n" << width << " " << height << "\n255\n";
	for (size_t pixel_index = 0; pixel_index < width * height; ++pixel_index) {
		for (size_t color_index = 0; color_index < 3; color_index++) {
			ofs << static_cast<char>(255 * std::max(0.f, std::min(1.f, buffer[pixel_index][color_index])));
		}
	}

	return 0;
}