#include "hip/hip_runtime.h"
//
// Created by andreas on 24.10.21.
//

// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include "cuda_playground/vector_add.cu"

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(int *__restrict__ a, int *__restrict__ b,
						  int *__restrict__ c, int N) {
	// Calculate global thread ID
	//int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	IntegerVectorAddition add;
	add.set_summand_one(a, N);
	add.set_summand_two(b, N);

	// Boundary check
	//if (tid < N) c[tid] = a[tid] + b[tid];
	add.add(c);
}

// Check vector add result
void verify_result(std::vector<int> &a, std::vector<int> &b,
				   std::vector<int> &c) {
	long long int sum = 0;
	for (int i = 0; i < a.size(); i++) {
		assert(c[i] == a[i] + b[i]);
		sum += c[i];
	}
	printf("Sum = %lld \n", sum);
}

int main() {
	// Array size of 2^16 (65536 elements)
	constexpr int N = 1 << 16;
	constexpr size_t bytes = sizeof(int) * N;

	// Vectors for holding the host-side (CPU-side) data
	std::vector<int> a;
	a.reserve(N);
	std::vector<int> b;
	b.reserve(N);
	std::vector<int> c;
	c.reserve(N);

	// Initialize random numbers in each array
	for (int i = 0; i < N; i++) {
		a.push_back(rand() % 100);
		b.push_back(rand() % 100);
	}

	// Allocate memory on the device
	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Copy data from the host to the device (CPU -> GPU)
	hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

	// Threads per CTA (1024)
	int NUM_THREADS = 1 << 10;

	// CTAs per Grid
	// We need to launch at LEAST as many threads as we have elements
	// This equation pads an extra CTA to the grid if N cannot evenly be divided
	// by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
	int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

	// Launch the kernel on the GPU
	// Kernel calls are asynchronous (the CPU program continues execution after
	// call, but no necessarily before the kernel finishes)
	vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

	// Copy sum vector from device to host
	// hipMemcpy is a synchronous operation, and waits for the prior kernel
	// launch to complete (both go to the default stream in this case).
	// Therefore, this hipMemcpy acts as both a memcpy and synchronization
	// barrier.
	hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

	// Check result for errors
	verify_result(a, b, c);

	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
}
